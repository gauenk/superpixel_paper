#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cuda/std/type_traits>
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <vector>


#define CUDA_NUM_THREADS 256
#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

template <typename scalar_t>
__global__ void eff_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> attn,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> samples,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> out,
    int batchsize, int num_superpixels, int psize, int nsamples,
    int pk_nthreads, int num_pk){

    // -- unpack --
    int bi = blockIdx.z;
    int spi = blockIdx.y;
    int ntotal = nsamples*psize*psize*pk_nthreads;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= ntotal){ return; }
    int si = index % nsamples;
    int tmp = index / (nsamples);
    int pi = tmp % psize;
    tmp = index / (nsamples*psize);
    int pj = tmp % psize;
    tmp = index / (nsamples*psize*psize);
    // int pk_init = tmp % pk_nthreads;
    int pk_init = 0;
    int PK = psize;//min(pk_init+num_pk,psize);
    if ((si >= nsamples) or (pi >= psize) or (pj >= psize)){ return; }

    scalar_t acc = 0;
    for (int pk=pk_init;pk<PK;pk++){
      // -- skip eqs --
      if (pk == pi){ continue; }
      if (pk == pj){ continue; }
      acc += attn[bi][spi][pi][pk]*samples[bi][spi][pk][si];
    }

    // -- init output --
    acc += attn[bi][spi][pi][pi];
    acc += attn[bi][spi][pi][pj];

    // -- accumulate average into (pi,pj) --
    atomicAdd(&out[bi][spi][pi][pj],acc/nsamples);

}

void eff_forward_cuda(
    const torch::Tensor attn,
    const torch::Tensor samples,
    torch::Tensor out){

    // -- check --
    CHECK_INPUT(attn);
    CHECK_INPUT(samples);
    CHECK_INPUT(out);

    // -- unpack --
    int batchsize = attn.size(0);
    int num_superpixels = attn.size(1);
    int psize = attn.size(2);
    int nsamples = samples.size(3);
    int num_pk = psize;

    // -- block --
    int pk_nthreads = (psize-1) / num_pk + 1;
    int ntotal = nsamples*psize*psize*pk_nthreads;
    dim3 block((ntotal-1)/CUDA_NUM_THREADS+1,num_superpixels,batchsize);
    AT_DISPATCH_FLOATING_TYPES(attn.type(), "forward_kernel", ([&] {
        eff_forward_kernel<scalar_t><<< block, CUDA_NUM_THREADS >>>(
            attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            samples.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            out.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            batchsize, num_superpixels, psize, nsamples, pk_nthreads, num_pk
        );
    }));

}



template <typename scalar_t>
__global__ void eff_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> attn_grad,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> normz_grad,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> normz,
    int batchsize, int num_superpixels, int psize){

    // -- unpack --
    int bi = blockIdx.z;
    int spi = blockIdx.y;
    int ntotal = psize*psize;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= ntotal){ return; }
    int pi = index % psize;
    int tmp = index / (psize);
    int pj = tmp % psize;
    int pk_init = 0;
    int PK = psize;
    if ((pi >= psize) or (pj >= psize)){ return; }

    scalar_t acc = 0;
    for (int pk=pk_init;pk<PK;pk++){
      // -- skip eqs --
      if (pk == pi){ continue; }
      if (pk == pj){ continue; }
      acc += normz_grad[bi][spi][pi][pi]*normz[bi][spi][pi][pi];
    }

    // -- init output --
    acc += normz_grad[bi][spi][pi][pi];
    acc += normz_grad[bi][spi][pi][pj];

    // -- accumulate average into (pi,pj) --
    atomicAdd(&attn_grad[bi][spi][pi][pj],acc);

}

void eff_backward_cuda(
    torch::Tensor attn_grad,
    const torch::Tensor normz_grad,
    const torch::Tensor normz){

    // -- check --
    CHECK_INPUT(attn_grad);
    CHECK_INPUT(normz_grad);
    CHECK_INPUT(normz);

    // -- unpack --
    int batchsize = attn_grad.size(0);
    int num_superpixels = attn_grad.size(1);
    int psize = attn_grad.size(2);

    // -- block --
    // int pk_nthreads = (psize-1) / num_pk + 1;
    int ntotal = psize*psize;
    dim3 block((ntotal-1)/CUDA_NUM_THREADS+1,num_superpixels,batchsize);
    AT_DISPATCH_FLOATING_TYPES(attn_grad.type(), "backward_kernel", ([&] {
        eff_backward_kernel<scalar_t><<< block, CUDA_NUM_THREADS >>>(
            attn_grad.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            normz_grad.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            normz.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            batchsize, num_superpixels, psize
        );
    }));

}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("eff_forward", &eff_forward_cuda, "efficient normz forward");
  m.def("eff_backward", &eff_backward_cuda, "efficient normz forward");
}

